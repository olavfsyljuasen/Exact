#include "hip/hip_runtime.h"
#ifndef CUDA_PRECISION_CU
#define CUDA_PRECISION_CU

//routines that calls cuda_kernels, i.e. driver-routines for cuda calls
//routines with prefix cu as in cuRoutine can be called directly from host code.


#ifdef DEBUG
const bool WTRACE=true;
#else
const bool WTRACE=false;
#endif

#ifdef DEBUG2
const bool TRACE=true;
#else
const bool TRACE=false;
#endif

#include<iostream>
#include<iomanip>

#include "cuda_global.h"
#include "cuda_precision.h"
#include "cuda_precision_kernels.h"

//#include "precision_types.h"


int cuda_mat_vec_multiply_cmplx(
const int M,const int Nc,const int* indices ,
const void* data , const cmplxformat* x, cmplxformat* y)
{
  if(WTRACE) cout << "Starting cuda_mat_vec_multiply_cmplx" << endl;
  int b = (M + THREADS_PER_BLOCK - 1)/THREADS_PER_BLOCK;
  
  const cuda_cmplx* datap=reinterpret_cast<const cuda_cmplx*>(data);
  const void* xp=reinterpret_cast<const void*>(x);
  void* yp=reinterpret_cast<void*>(y);

  cuda_kernel_mat_vec_multiply_cmplx<<<b, THREADS_PER_BLOCK>>>(M,Nc,indices,datap,xp,yp);

  if(WTRACE) cout << "Done with cuda_mat_vec_multiply_cmplx" << endl;
  return hipSuccess;
}

int cuda_real_Zmaxpy(const int N,const realformat* q,cmplxformat* d_x,const cmplxformat* d_y) 
{
  if(WTRACE) cout << "Starting cuda_real_Zmaxpy" << endl;

  realformat* d_q=NULL;
  AllocateSpaceOnDevice(&d_q,1);
  UploadToDevice(q,1,&d_q);

  int b = (N + THREADS_PER_BLOCK - 1)/THREADS_PER_BLOCK;
  cuda_kernel_real_Zmaxpy<<<b, THREADS_PER_BLOCK>>>(N,d_q,d_x,d_y);

  FreeMemoryOnDevice(&d_q);

  if(WTRACE) cout << "Done with cuda_real_Zmaxpy" << endl;
  return hipSuccess;
}

int cuda_cmplx_Zmaxpy(const int N,const realformat* q,void* d_x,const void* d_y) 
{
  if(WTRACE) cout << "Starting cuda_cmplx_Zmaxpy" << endl;

  realformat* d_q=NULL;
  AllocateSpaceOnDevice(&d_q,1);
  UploadToDevice(q,1,&d_q);

  int b = (N + THREADS_PER_BLOCK - 1)/THREADS_PER_BLOCK;
  cuda_kernel_cmplx_Zmaxpy<<<b, THREADS_PER_BLOCK>>>(N,d_q,d_x,d_y);

  FreeMemoryOnDevice(&d_q);

  if(WTRACE) cout << "Done with cuda_cmplx_Zmaxpy" << endl;
  return hipSuccess;
}


int cuda_realdotproduct_real(const int N,const realformat* d_x,const realformat* d_y,realformat* ans) 
{

  if(WTRACE) cout << "Starting cuda_realdotproduct_cmplx" << endl;
  hipError_t err;

  realformat c[BLOCKS];
  for(int i=0; i<BLOCKS; i++){ c[i]=realformat(0.);}
  realformat* d_c=NULL;

  AllocateSpaceOnDevice(&d_c,BLOCKS);
  UploadToDevice(c,BLOCKS,&d_c);

  cuda_kernel_realdotproduct_real<<<BLOCKS, THREADS_PER_BLOCK>>>(N, d_x, d_y, d_c); 

  err = hipGetLastError();
  if(err != hipSuccess)
    {
      fprintf(stderr, "Failed to execute (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }
  
  DownloadFromDevice(&d_c,BLOCKS,c);
  FreeMemoryOnDevice(&d_c);

  // sum the contribution from all the blocks:
  unsigned int old_cw;
  fpu_fix_start(&old_cw);
  for(int i = 1; i < BLOCKS; i++){c[0] += c[i];}
  fpu_fix_end(&old_cw); 

  *ans=c[0];

  if(WTRACE) cout << "Done with cuda_realdotproduct_cmplx" << endl;
  return hipSuccess;
}



int cuda_realdotproduct_cmplx(const int N,const cmplxformat* d_x,const cmplxformat* d_y,realformat* ans) 
{

  if(WTRACE) cout << "Starting cuda_realdotproduct_cmplx" << endl;
  hipError_t err;

  realformat c[BLOCKS];
  for(int i=0; i<BLOCKS; i++){ c[i]=realformat(0.);}
  realformat* d_c=NULL;

  AllocateSpaceOnDevice(&d_c,BLOCKS);
  UploadToDevice(c,BLOCKS,&d_c);

  cuda_kernel_realdotproduct_cmplx<<<BLOCKS, THREADS_PER_BLOCK>>>(N, d_x, d_y, d_c); 

  err = hipGetLastError();
  if(err != hipSuccess)
    {
      fprintf(stderr, "Failed to execute (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }
  
  DownloadFromDevice(&d_c,BLOCKS,c);
  FreeMemoryOnDevice(&d_c);

  // sum the contribution from all the blocks:
  unsigned int old_cw;
  fpu_fix_start(&old_cw);
  for(int i = 1; i < BLOCKS; i++){c[0] += c[i];}
  fpu_fix_end(&old_cw); 


  *ans=c[0];

  if(WTRACE) cout << "Done with cuda_realdotproduct_cmplx" << endl;
  return hipSuccess;
}


int cuda_cmplxdotproduct_cmplx(const int N,const cmplxformat* d_x,const cmplxformat* d_y,cmplxformat* ans) 
{
  if(WTRACE) cout << "Starting cuda_cmplxdotproduct_cmplx" << endl;
  hipError_t err;
  
  cmplxformat c[BLOCKS];
  for(int i=0; i<BLOCKS; i++){ c[i]=cmplxformat(0.,0.);}
  cmplxformat* d_c=NULL;
  
  AllocateSpaceOnDevice(&d_c,BLOCKS);
  UploadToDevice(c,BLOCKS,&d_c);

  cuda_kernel_cmplxdotproduct_cmplx<<<BLOCKS, THREADS_PER_BLOCK>>>(N, d_x, d_y, d_c); 

  err = hipGetLastError();
  if(err != hipSuccess)
    {
      fprintf(stderr, "Failed to execute (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }
  
  DownloadFromDevice(&d_c,BLOCKS,c);
  FreeMemoryOnDevice(&d_c);

  // sum the contribution from all the blocks:
  unsigned int old_cw;
  fpu_fix_start(&old_cw);
  for(int i = 1; i < BLOCKS; i++){c[0].x += c[i].x; c[0].y += c[i].y;}
  fpu_fix_end(&old_cw); 

  *ans=c[0];

  if(WTRACE) cout << "Done with cuda_cmplxdotproduct_cmplx" << endl;
  return hipSuccess;
}

// computes the dot product between two real vectors
int cuRealDotProductReal(const int N,const double* x,const double* y,double* res)
{
  realformat* d_x=NULL;
  AllocateSpaceOnDevice(&d_x,N);
  UploadToDeviceAndExpand(x,N,&d_x);

  realformat* d_y=NULL;
  AllocateSpaceOnDevice(&d_y,N);
  UploadToDeviceAndExpand(y,N,&d_y);

  realformat rp =realformat(0.);

  cuda_realdotproduct_real(N,d_x,d_y,&rp); 

  *res=todouble(rp);

  FreeMemoryOnDevice(&d_x);
  FreeMemoryOnDevice(&d_y);
  
  return 0;
}

// computes the real part of the dot product between two complex vectors
int cuRealDotProductCmplx(const int N,const void* x,const void* y,double* res)
{
  cmplxformat* d_x=NULL;
  AllocateSpaceOnDevice(&d_x,N);
  UploadToDeviceAndExpand(x,N,&d_x);  

  cmplxformat* d_y=NULL;
  AllocateSpaceOnDevice(&d_y,N);
  UploadToDeviceAndExpand(y,N,&d_y);  

  realformat rp =realformat(0.);

  cuda_realdotproduct_cmplx(N,d_x,d_y,&rp); 

  *res=todouble(rp);

  FreeMemoryOnDevice(&d_x);
  FreeMemoryOnDevice(&d_y);

  return 0;
}

/*
int cuConvertToDouble(void* p,double* d)
{
  realformat pval=reinterpret_cast<realformat>(*p);
  *d = todouble(pval);
}
*/

// computes the dot product between two complex vectors
int cuCmplxDotProductCmplx(int N,void* x,void* y,void* res)
{
  cmplxformat* d_x=NULL;
  AllocateSpaceOnDevice(&d_x,N);
  UploadToDeviceAndExpand(x,N,&d_x);  

  cmplxformat* d_y=NULL;
  AllocateSpaceOnDevice(&d_y,N);
  UploadToDeviceAndExpand(y,N,&d_y);  

  cmplxformat rp =cmplxformat(0.,0.);

  cuda_cmplxdotproduct_cmplx(N,d_x,d_y,&rp); 

  cuda_cmplx* res_ptr=reinterpret_cast<cuda_cmplx*>(res);

  (*res_ptr).x=todouble(rp.x);
  (*res_ptr).y=todouble(rp.y);

  FreeMemoryOnDevice(&d_x);
  FreeMemoryOnDevice(&d_y);
  
  return 0;
}


int cuda_norm_cmplx(const int N,const cmplxformat* d_x,void* ans) 
{

  if(WTRACE) cout << "Starting cuda_norm_cmplx" << endl;
  hipError_t err;

  realformat c[BLOCKS];
  for(int i=0; i<BLOCKS; i++){ c[i]=realformat(0.);}
  realformat* d_c=NULL;

  AllocateSpaceOnDevice(&d_c,BLOCKS);
  UploadToDevice(&c[0],BLOCKS,&d_c);

  cuda_kernel_realdotproduct_cmplx<<<BLOCKS, THREADS_PER_BLOCK>>>(N, d_x, d_x, d_c); 

  err = hipGetLastError();
  if(err != hipSuccess)
    {
      fprintf(stderr, "Failed to execute (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }
  
  DownloadFromDevice(&d_c,BLOCKS,c);
  FreeMemoryOnDevice(&d_c);

  // sum the contribution from all the blocks:
  unsigned int old_cw;
  fpu_fix_start(&old_cw);
  for(int i = 1; i < BLOCKS; i++){c[0] += c[i];}
  fpu_fix_end(&old_cw); 

  realformat* ansp = reinterpret_cast<realformat*>(ans);
  *ansp=sqrt(c[0]);

  if(WTRACE) cout << "Done with cuda_norm_cmplx" << endl;
  return hipSuccess;
}

// divide the complex vector x by  a real number: x=x/q
int cuda_Zdinvscal(int N,realformat* q,cmplxformat* d_x)
{
  if(WTRACE) cout << "Starting cuda_Zdinvscal " << endl;

  realformat s= inv(*q); // s= 1/q taking the inverse
  
  realformat* d_s=NULL;
  AllocateSpaceOnDevice(&d_s,1);
  UploadToDevice(&s,1,&d_s);

  int b = (N + THREADS_PER_BLOCK - 1)/THREADS_PER_BLOCK;
  cuda_kernel_Zdscal<<<b, THREADS_PER_BLOCK>>>(N,d_s,d_x);

  FreeMemoryOnDevice(&d_s);

  if(WTRACE) cout << "Done with cuda_Zdinvscal" << endl;
  return hipSuccess;
}


// multiply the complex vector x by  a real number: x=x*q
int cuda_Zdscal(int N,realformat* q,cmplxformat* d_x)
{
  if(WTRACE) cout << "Starting cuda_Zdscal " << endl;

  realformat* d_s=NULL;
  AllocateSpaceOnDevice(&d_s,1);
  UploadToDevice(q,1,&d_s);

  int b = (N + THREADS_PER_BLOCK - 1)/THREADS_PER_BLOCK;
  cuda_kernel_Zdscal<<<b, THREADS_PER_BLOCK>>>(N,d_s,d_x);

  FreeMemoryOnDevice(&d_s);

  if(WTRACE) cout << "Done with cuda_Zdscal" << endl;
  return hipSuccess;
}


int cuda_Normalize(const int N,cmplxformat* d_x,double* ans) 
{
  if(WTRACE) cout << "Starting cuda_Normalize" << endl;
  realformat norm=makereal(0.);
  cuda_norm_cmplx(N,d_x,&norm); 
  cuda_Zdinvscal(N,&norm,d_x);
  *ans=todouble(norm);
  return hipSuccess;  
}


int AllocateSpaceOnDevice(cmplxformat** d_p,const int N)
{
  if(WTRACE) cout << "In AllocateSpaceOnDevice" << endl;	

  if (hipMalloc(d_p, N*sizeof(cmplxformat)) != hipSuccess)
    {
      fprintf(stderr, "!device memory allocatioNerror (AllocateSpaceOnDevice)\n");
      return EXIT_FAILURE;
    }
  return hipSuccess;
}

int AllocateSpaceOnDevice(realformat** d_p,const int N)
{
  if(WTRACE) cout << "In AllocateSpaceOnDevice" << endl;	

  if (hipMalloc(d_p, N*sizeof(realformat)) != hipSuccess)
    {
      fprintf(stderr, "!!!! device memory allocatioNerror (AllocateSpaceOnDevice)\n");
      return EXIT_FAILURE;
    }
  return hipSuccess;
}

int UploadToDevice(const cmplxformat* phi,const int N,cmplxformat** d_phi_ptr)
{
  if(WTRACE) cout << "In UploadToDevice" << endl;	
  hipError_t err;

  err = hipMemcpy(*d_phi_ptr, phi, N*sizeof(cmplxformat), hipMemcpyHostToDevice);

  if(err != hipSuccess )
    {
      fprintf(stderr, "Failed to copy from host to device (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }
  return hipSuccess;
}

int UploadToDevice(const realformat* phi,const int N,realformat** d_phi_ptr)
{
  if(WTRACE) cout << "In UploadToDevice" << endl;	
  hipError_t err;

  err = hipMemcpy(*d_phi_ptr, phi, N*sizeof(realformat), hipMemcpyHostToDevice);

  if(err != hipSuccess )
    {
      fprintf(stderr, "Failed to copy from host to device (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }
  return hipSuccess;
}


int DownloadFromDevice(cmplxformat **d_phi_ptr,const int N,cmplxformat* phi)
{
  if(WTRACE) cout << "In DownloadFromDevice" << endl;	
  hipError_t err;

  err = hipMemcpy(phi,*d_phi_ptr,N*sizeof(cmplxformat),hipMemcpyDeviceToHost);

  if(err != hipSuccess )
    {
      fprintf(stderr, "Failed to copy from device to host (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }
  return hipSuccess;
}

int DownloadFromDevice(realformat **d_phi_ptr,const int N,realformat* phi)
{
  if(WTRACE) cout << "In DownloadFromDevice" << endl;	
  hipError_t err;

  err = hipMemcpy(phi,*d_phi_ptr, N*sizeof(realformat), hipMemcpyDeviceToHost);

  if(err != hipSuccess )
    {
      fprintf(stderr, "Failed to copy from device to host (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }
  return hipSuccess;
}

int FreeMemoryOnDevice(cmplxformat** d_phi)
{
  if(WTRACE) cout << "In FreeMemoryOnDevice" << endl;	
  hipError_t err;

  err = hipFree(*d_phi);
  if (err != hipSuccess)
    {
      fprintf(stderr, "Failed to free memory on device (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }
  return hipSuccess;
}

int FreeMemoryOnDevice(realformat** d_phi)
{
  if(WTRACE) cout << "In FreeMemoryOnDevice" << endl;	
  hipError_t err;

  err = hipFree(*d_phi);
  if (err != hipSuccess)
    {
      fprintf(stderr, "Failed to free memory on device (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }
  return hipSuccess;
}



int UploadToDeviceAndExpand(const void* p,const int N,cmplxformat** d_p)
{
  const cuda_cmplx* pptr=reinterpret_cast<const cuda_cmplx*>(p);
  cmplxformat cp[N];
  for(int i=0; i<N; i++){ cp[i]=cmplxformat(pptr[i].x,pptr[i].y);}
  UploadToDevice(cp,N,d_p);
  return hipSuccess;
}

int UploadToDeviceAndExpand(const double* p,const int N,realformat** d_p)
{
  realformat cp[N];
  for(int i=0; i<N; i++){ cp[i]=realformat(p[i]);}
  UploadToDevice(cp,N,d_p);
  return hipSuccess;
}

int DownloadFromDeviceAndContract(cmplxformat** d_p,int N,void* p)
{
  cuda_cmplx* pptr=reinterpret_cast<cuda_cmplx*>(p);
  cmplxformat cp[N];
  DownloadFromDevice(d_p,N,cp);

  for(int i=0; i<N; i++){ pptr[i]=make_hipDoubleComplex(todouble(cp[i].x),todouble(cp[i].y));}
  return hipSuccess;
}

int DownloadFromDeviceAndContract(realformat** d_p,int N,double* p)
{
  realformat cp[N];
  DownloadFromDevice(d_p,N,cp);

  for(int i=0; i<N; i++){ p[i]=todouble(cp[i]);}
  return hipSuccess;
}

int InspectDevice(cmplxformat** d_p,int N)
{
  cuda_cmplx cp[N];
  DownloadFromDeviceAndContract(d_p,N,cp);

  for(int i=0; i<N; i++){cout << cp[i] << endl;}
  return hipSuccess;
}

int InspectDevice(realformat** d_p,int N)
{
  double cp[N];
  DownloadFromDeviceAndContract(d_p,N,cp);

  for(int i=0; i<N; i++){cout << cp[i] << endl;}
  return hipSuccess;
}


/*
//*****************************************************************

int cuda_Zaxpy(int N,cuda_cmplx* q,cuda_cmplx* x,cuda_cmplx* y)
{
  hipError_t err;
  if(WTRACE) cout << "In cuda_Zaxpy" << endl;

  cuda_cmplx* d_q=NULL;
  AllocateSpaceOnDevice(1,&d_q);

  UploadToDevice(q,1,&d_q);

  int  blockspergrid = (N + THREADS_PER_BLOCK - 1)/THREADS_PER_BLOCK;
  if(WTRACE) cout << "Launcing cuda_Zaxpy with " << blockspergrid << " blocks" << endl;
#ifdef DOUBLEDOUBLE
  cudakernel_Zaxpy_dd<<<blockspergrid, THREADS_PER_BLOCK>>>(N,d_q,x,y); 
#elif QUADDOUBLE
  cudakernel_Zaxpy_qd<<<blockspergrid, THREADS_PER_BLOCK>>>(N,d_q,x,y); 
#else
  cudakernel_Zaxpy<<<blockspergrid, THREADS_PER_BLOCK>>>(N,d_q,x,y); 
#endif      

  err = hipGetLastError();
  if(err != hipSuccess)
    {
      fprintf(stderr, "Failed to execute cuda_Zaxpy (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

  FreeMemoryOnDevice(&d_q);

  if(WTRACE) cout << "Done with cuda_Zaxpy" << endl;
  return hipSuccess;
}


int cuZaxpy(int N,void* q,void* x,void* y)
{
  if(WTRACE) cout << "Starting Zaxpy " << endl;
  cuda_cmplx* d_x=NULL;
  AllocateSpaceOnDevice(N,&d_x);
  
  cuda_cmplx* x_ptr=reinterpret_cast<cuda_cmplx*>(x);
  UploadToDevice(x_ptr,N,&d_x);

  cuda_cmplx* d_y=NULL;
  AllocateSpaceOnDevice(N,&d_y);
  
  cuda_cmplx* y_ptr=reinterpret_cast<cuda_cmplx*>(y);
  UploadToDevice(y_ptr,N,&d_y);

  cuda_cmplx* q_ptr=reinterpret_cast<cuda_cmplx*>(q);

  cuda_Zaxpy(N,q_ptr,d_x,d_y);

  DownloadFromDevice(&d_x,N,x_ptr);

  FreeMemoryOnDevice(&d_x);
  FreeMemoryOnDevice(&d_y);
  
  if(WTRACE) cout << "Done with Zaxpy " << endl;

  return hipSuccess;
}




int cuda_ComplexDotProduct(int N,cuda_cmplx* x,cuda_cmplx* y,cuda_cmplx* res)
{
  if(WTRACE) cout << "Starting cuda_ComplexDotProduct" << endl;
  hipError_t err;

#ifdef DOUBLEDOUBLE
  dd_real cr[BLOCKS];
  dd_real ci[BLOCKS];
  for(int i=0; i<BLOCKS; i++){ cr[i]=dd_real(0.); ci[i]=dd_real(0.);}
  void* d_cr=NULL;
  void* d_ci=NULL;
  int sizeofdouble=sizeof(dd_real);
#elif QUADDOUBLE
  qd_real cr[BLOCKS];
  qd_real ci[BLOCKS];
  for(int i=0; i<BLOCKS; i++){ cr[i]=qd_real(0.); ci[i]=qd_real(0.);}
  void* d_cr=NULL;
  void* d_ci=NULL;
  int sizeofdouble=sizeof(qd_real);
#else
  double cr[BLOCKS];
  double ci[BLOCKS];
  for(int i=0; i<BLOCKS; i++){ cr[i]=0.; ci[i]=0.;}
  void* d_cr=NULL;
  void* d_ci=NULL;
  int sizeofdouble=sizeof(double);
#endif

  err = hipMalloc(&d_cr, sizeofdouble * BLOCKS);
  err = hipMalloc(&d_ci, sizeofdouble * BLOCKS);
  if(err !=hipSuccess)
    {
      fprintf(stderr, "!!!! device memory allocatioNerror (ComplexDotProduct)\n");
      return EXIT_FAILURE;
    }

  err=hipMemcpy(d_cr,&cr[0], sizeofdouble*BLOCKS, hipMemcpyHostToDevice);
  err=hipMemcpy(d_ci,&ci[0], sizeofdouble*BLOCKS, hipMemcpyHostToDevice);
  if(err != hipSuccess) 
    {
      fprintf(stderr, "Failed to copy from host to device (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }
#ifdef DOUBLEDOUBLE
  cudakernel_dotproduct_cmplx_dd<<<BLOCKS, THREADS_PER_BLOCK>>>(N, x, y, d_cr, d_ci); 
#elif QUADDOUBLE
  cudakernel_dotproduct_cmplx_qd<<<BLOCKS, THREADS_PER_BLOCK>>>(N, x, y, d_cr, d_ci); 
#else
  cudakernel_dotproduct_cmplx<<<BLOCKS, THREADS_PER_BLOCK>>>(N, x, y, d_cr, d_ci); 
#endif  

  err = hipGetLastError();
  if(err != hipSuccess)
    {
      fprintf(stderr, "Failed to execute (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }
  
  err=hipMemcpy(&cr[0],d_cr,sizeofdouble*BLOCKS, hipMemcpyDeviceToHost);
  err=hipMemcpy(&ci[0],d_ci,sizeofdouble*BLOCKS, hipMemcpyDeviceToHost);
  if(err != hipSuccess) 
    {
      fprintf(stderr, "Failed to copy from device to host (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

  err=hipFree(d_cr);
  err=hipFree(d_ci);
  if ( err != hipSuccess)
    {
      fprintf(stderr, "Failed to free memory on device (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

  unsigned int old_cw;
  fpu_fix_start(&old_cw);

  for(int i = 1; i < BLOCKS; i++)
    { 
      cr[0] += cr[i];
      ci[0] += ci[i];
    }

#if defined DOUBLEDOUBLE || defined QUADDOUBLE  
  *res=make_hipDoubleComplex(to_double(cr[0]),to_double(ci[0]));
#else
  *res=make_hipDoubleComplex(cr[0],ci[0]);
#endif

  fpu_fix_end(&old_cw); 
  if(WTRACE) cout << "Done with cuda_ComplexDotProduct" << endl;
  return hipSuccess;
} 



int cuComplexDotProduct(int N,void* x,void* y,void* res)
{
  cuda_cmplx* d_x=NULL;
  AllocateSpaceOnDevice(N,&d_x);
  
  cuda_cmplx* x_ptr=reinterpret_cast<cuda_cmplx*>(x);
  UploadToDevice(x_ptr,N,&d_x);

  cuda_cmplx* d_y=NULL;
  AllocateSpaceOnDevice(N,&d_y);
  
  cuda_cmplx* y_ptr=reinterpret_cast<cuda_cmplx*>(y);
  UploadToDevice(y_ptr,N,&d_y);

  cuda_cmplx* res_ptr=reinterpret_cast<cuda_cmplx*>(res);
  
  cuda_ComplexDotProduct(N,d_x,d_y,res_ptr);

  FreeMemoryOnDevice(&d_x);
  FreeMemoryOnDevice(&d_y);
  
  return 0;
}


//***********************************************************************
// cuda norm routines

int cuda_Dznrm2(int N,cuda_cmplx* d_x,double* res)
{
  if(WTRACE) cout << "Starting cuda_Dznrm2" << endl;
  hipError_t err;

#ifdef DOUBLEDOUBLE
  dd_real c[BLOCKS];
  for(int i=0; i<BLOCKS; i++){ c[i]=dd_real(0.);}
  void* d_c=NULL;
  int sizeofdouble=sizeof(dd_real);
#elif QUADDOUBLE
  qd_real c[BLOCKS];
  for(int i=0; i<BLOCKS; i++){ c[i]=qd_real(0.);}
  void* d_c=NULL;
  int sizeofdouble=sizeof(qd_real);
#else
  double c[BLOCKS];
  for(int i=0; i<BLOCKS; i++){ c[i]=0.;}
  void* d_c=NULL;
  int sizeofdouble=sizeof(double);
#endif

  err = hipMalloc(&d_c, sizeofdouble * BLOCKS);
  if(err !=hipSuccess)
    {
      fprintf(stderr, "!!!! device memory allocatioNerror (ComplexDotProduct)\n");
      return EXIT_FAILURE;
    }

  err=hipMemcpy(d_c,&c[0], sizeofdouble*BLOCKS, hipMemcpyHostToDevice);
  if(err != hipSuccess) 
    {
      fprintf(stderr, "Failed to copy from host to device (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }
#ifdef DOUBLEDOUBLE
  cudakernel_norm_cmplx_dd<<<BLOCKS, THREADS_PER_BLOCK>>>(N, d_x, d_c); 
#elif QUADDOUBLE
  cudakernel_norm_cmplx_qd<<<BLOCKS, THREADS_PER_BLOCK>>>(N, d_x, d_c); 
#else
  cudakernel_norm_cmplx<<<BLOCKS, THREADS_PER_BLOCK>>>(N, d_x, d_c); 
#endif  

  err = hipGetLastError();
  if(err != hipSuccess)
    {
      fprintf(stderr, "Failed to execute (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }
  
  err=hipMemcpy(&c[0],d_c,sizeofdouble*BLOCKS, hipMemcpyDeviceToHost);
  if(err != hipSuccess) 
    {
      fprintf(stderr, "Failed to copy from device to host (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

  err=hipFree(d_c);
  if ( err != hipSuccess)
    {
      fprintf(stderr, "Failed to free memory on device (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

  unsigned int old_cw;
  fpu_fix_start(&old_cw);

  for(int i = 1; i < BLOCKS; i++)
    { 
      c[0] += c[i];
    }

#if defined DOUBLEDOUBLE || defined QUADDOUBLE 
  *res=to_double(sqrt(c[0])); 
#else
  *res=sqrt(c[0]);
#endif

  fpu_fix_end(&old_cw);
  if(WTRACE) cout << "Done with cuda_Dznrm2" << endl;
  return hipSuccess;
} 



int cuDznrm2(int N,void* x,double* res)
{
  cuda_cmplx* d_x=NULL;
  AllocateSpaceOnDevice(N,&d_x);
  
  cuda_cmplx* x_ptr=reinterpret_cast<cuda_cmplx*>(x);
  UploadToDevice(x_ptr,N,&d_x);

  cuda_Dznrm2(N,d_x,res);

  FreeMemoryOnDevice(&d_x);
  
  return 0;
}

//*************************************************************************************
// Normalize a complex vector

int cuda_Normalize(int N,cuda_cmplx* d_x,double* res)
{
  if(WTRACE) cout << "Starting cuda_Dznrm2" << endl;
  hipError_t err;

#ifdef DOUBLEDOUBLE
  dd_real c[BLOCKS];
  for(int i=0; i<BLOCKS; i++){ c[i]=dd_real(0.);}
  void* d_c=NULL;
  dd_real norm=dd_real(0.);
  dd_real invnorm=dd_real(0.);
  int sizeofdouble=sizeof(dd_real);
#elif QUADDOUBLE
  qd_real c[BLOCKS];
  for(int i=0; i<BLOCKS; i++){ c[i]=qd_real(0.);}
  void* d_c=NULL;
  qd_real norm=qd_real(0.);
  qd_real invnorm=qd_real(0.);
  int sizeofdouble=sizeof(qd_real);
#else
  double c[BLOCKS];
  for(int i=0; i<BLOCKS; i++){ c[i]=0.;}
  void* d_c=NULL;
  double norm=0.;
  double invnorm=0.;
  int sizeofdouble=sizeof(double);
#endif

  err = hipMalloc(&d_c, sizeofdouble * BLOCKS);
  if(err !=hipSuccess)
    {
      fprintf(stderr, "!!!! device memory allocatioNerror (ComplexDotProduct)\n");
      return EXIT_FAILURE;
    }

  err=hipMemcpy(d_c,&c[0], sizeofdouble*BLOCKS, hipMemcpyHostToDevice);
  if(err != hipSuccess) 
    {
      fprintf(stderr, "Failed to copy from host to device (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }
#ifdef DOUBLEDOUBLE
  cudakernel_norm_cmplx_dd<<<BLOCKS, THREADS_PER_BLOCK>>>(N, d_x, d_c); 
#elif QUADDOUBLE
  cudakernel_norm_cmplx_qd<<<BLOCKS, THREADS_PER_BLOCK>>>(N, d_x, d_c); 
#else
  cudakernel_norm_cmplx<<<BLOCKS, THREADS_PER_BLOCK>>>(N, d_x, d_c); 
#endif  

  err = hipGetLastError();
  if(err != hipSuccess)
    {
      fprintf(stderr, "Failed to execute (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }
  
  err=hipMemcpy(&c[0],d_c,sizeofdouble*BLOCKS, hipMemcpyDeviceToHost);
  if(err != hipSuccess) 
    {
      fprintf(stderr, "Failed to copy from device to host (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

  unsigned int old_cw;
  fpu_fix_start(&old_cw);

  for(int i = 1; i < BLOCKS; i++)
    { 
      c[0] += c[i];
    }

  norm=sqrt(c[0]);

#if defined DOUBLEDOUBLE || defined QUADDOUBLE
  *res=to_double(norm);
#else
  *res=norm;
#endif
  
  // use c[0] to store the inverse norm and then multiply by c[0]
  c[0]=inv(norm);

  fpu_fix_end(&old_cw);

  
  err=hipMemcpy(d_c,&c[0], sizeofdouble*1, hipMemcpyHostToDevice);
  if(err != hipSuccess) 
    {
      fprintf(stderr, "Failed to copy from host to device (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

  int  blockspergrid = (N + THREADS_PER_BLOCK - 1)/THREADS_PER_BLOCK;
#ifdef DOUBLEDOUBLE
  cudakernel_Zddscal_dd<<<blockspergrid, THREADS_PER_BLOCK>>>(N,d_c,d_x); 
#elif QUADDOUBLE
  cudakernel_Zqdscal_qd<<<blockspergrid, THREADS_PER_BLOCK>>>(N,d_c,d_x); 
#else
  cudakernel_Zdscal<<<blockspergrid, THREADS_PER_BLOCK>>>(N,d_c,d_x); 
#endif      

  err=hipFree(d_c);
  if ( err != hipSuccess)
    {
      fprintf(stderr, "Failed to free memory on device (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

  
  if(WTRACE) cout << "Done with cuda_norm_driver" << endl;
  return hipSuccess;
} 


int cuNormalize(int N,void* x,double* res)
{
  cuda_cmplx* d_x=NULL;
  AllocateSpaceOnDevice(N,&d_x);
  
  cuda_cmplx* x_ptr=reinterpret_cast<cuda_cmplx*>(x);
  UploadToDevice(x_ptr,N,&d_x);

  cuda_Normalize(N,d_x,res);

  DownloadFromDevice(&d_x,N,x_ptr);

  FreeMemoryOnDevice(&d_x);
  
  return 0;
}


//***********************  the double dot product: *************************************


int cuda_DoubleDotProduct(int N,double* d_x,double* d_y,double* res)
{
  hipError_t err;

#ifdef DOUBLEDOUBLE
  dd_real c[BLOCKS];
  for(int i=0; i<BLOCKS; i++) c[i]=dd_real(0.);
  void* d_c=NULL;
  int sizeofdouble=sizeof(dd_real);
#elif QUADDOUBLE
  qd_real c[BLOCKS];
  for(int i=0; i<BLOCKS; i++) c[i]=qd_real(0.);
  void* d_c=NULL;
  int sizeofdouble=sizeof(qd_real);
#else
  double c[BLOCKS];
  for(int i=0; i<BLOCKS; i++) c[i]=0.;
  void* d_c=NULL;
  int sizeofdouble=sizeof(double);
#endif

  err = hipMalloc(&d_c, sizeofdouble * BLOCKS);
  if(err !=hipSuccess)
    {
      fprintf(stderr, "!!!! device memory allocatioNerror (DoubleDotProduct)\n");
      return EXIT_FAILURE;
    }

  err=hipMemcpy(d_c,&c[0], sizeofdouble*BLOCKS, hipMemcpyHostToDevice);
  if(err != hipSuccess) 
    {
      fprintf(stderr, "Failed to copy from host to device (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

#ifdef DOUBLEDOUBLE
  cudakernel_dotproduct_double_dd<<<BLOCKS, THREADS_PER_BLOCK>>>(N, d_x, d_y, d_c); 
#elif QUADDOUBLE
  cudakernel_dotproduct_double_qd<<<BLOCKS, THREADS_PER_BLOCK>>>(N, d_x, d_y, d_c); 
#else
  cudakernel_dotproduct_double<<<BLOCKS, THREADS_PER_BLOCK>>>(N, d_x, d_y, d_c); 
#endif

  err = hipGetLastError();
  if(err != hipSuccess)
    {
      fprintf(stderr, "Failed to execute cuda_dotproduct_double_dd (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }
  
  err=hipMemcpy(&c[0],d_c,sizeofdouble*BLOCKS, hipMemcpyDeviceToHost);
  if(err != hipSuccess) 
    {
      fprintf(stderr, "Failed to copy from device to host (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

  err=hipFree(d_c);
  if ( err != hipSuccess)
    {
      fprintf(stderr, "Failed to free memory d_c on device (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

  unsigned int old_cw;
  fpu_fix_start(&old_cw);

  for(int i = 1; i < BLOCKS; i++){ c[0] += c[i];}

#if defined DOUBLEDOUBLE || defined QUADDOUBLE    
  *res=to_double(c[0]);
#else
  *res=c[0];
#endif

  fpu_fix_end(&old_cw);
  return hipSuccess;
} 
  

int cuDoubleDotProduct(int N,double* x,double* y,double* res)
{
  double* d_x=NULL;
  AllocateSpaceOnDevice(N,&d_x);
  
  UploadToDevice(x,N,&d_x);

  double* d_y=NULL;
  AllocateSpaceOnDevice(N,&d_y);
  
  UploadToDevice(y,N,&d_y);

  cuda_DoubleDotProduct(N,d_x,d_y,res);

  FreeMemoryOnDevice(&d_x);
  FreeMemoryOnDevice(&d_y);
  
  return hipSuccess;
}
*/


#endif // CUDA_PRECISION_DRIVERS_CU
